#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <iostream>
#include <serde-helper/serde.h>
#include "common.h"

namespace float_tokenization
{
/*std::uint32_t float_to_token(float input)
{
    union
    {
        float in;
        std::int32_t out;
    } data;
    data.in = input;
    std::bitset<sizeof(float) * CHAR_BIT> bits(data.out);
    const unsigned int mantissa_bits_to_truncate = 10;
    unsigned int token_exponent = 0;
    std::uint32_t token = 0;
    for(unsigned int i = 31; i > mantissa_bits_to_truncate; i--)
    {
        if(bits[i])
        {
            token += (1 << token_exponent);
        }
        token_exponent++;
    }
    return token;
}

float token_to_float(std::uint32_t input)
{
    std::bitset<sizeof(std::uint32_t) * CHAR_BIT> token_bits(input);
    std::bitset<sizeof(float) * CHAR_BIT> float_bits(0.0f);
    unsigned int float_bits_offset = 0;
    for(int i = 31; i > -1; i--)
    {
        float_bits[float_bits_offset++] = token_bits[i];
    }
    float output;
    memcpy(&output, &float_bits, sizeof(float));
    return output;
}*/

__global__ void tokenization_kernel(
    __half* input, uint32_t* output, 
    const uint32_t mantissa_bits_to_truncate, const uint32_t n_tokens
){
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid > 0) { return; }
    for(uint32_t i = 0; i < n_tokens; i++)
    {
        uint16_t bits = __half_as_ushort(input[i]);
        output[i] = bits >> mantissa_bits_to_truncate;
    }
}

__global__ void detokenization_kernel(
    uint32_t* input, __half* output,
    const uint32_t mantissa_bits_to_restore, const uint32_t n_tokens
){
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid > 0) { return; }
    for(uint32_t i = 0; i < n_tokens; i++)
    {
        uint16_t bits = static_cast<uint16_t>(input[i]);
        output[i] = __ushort_as_half(bits << mantissa_bits_to_restore);
    }
}

void launch_tokenization(
    hipStream_t stream, void** buffers, const char* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc = 
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);
    __half* input = static_cast<__half*>(buffers[0]);
    uint32_t* output = static_cast<uint32_t*>(buffers[1]);
    
    tokenization_kernel<<<1, 32, 0, stream>>>(
        input, output, 
        desc.mantissa_shift, desc.n_tokens
    );
}

void launch_detokenization(
    hipStream_t stream, void** buffers, const char* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc = 
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);
    uint32_t* input = static_cast<uint32_t*>(buffers[0]);
    __half* output = static_cast<__half*>(buffers[1]);
    
    detokenization_kernel<<<1, 32, 0, stream>>>(
        input, output, 
        desc.mantissa_shift, desc.n_tokens
    );
}

} // namespace float_tokenization
