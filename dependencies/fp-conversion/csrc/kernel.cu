//#include <hip/hip_fp16.h>
#include <iostream>
#include <serde-helper/serde.h>
#include "common.h"

namespace fp_conversion
{
/*std::uint32_t float_to_token(float input)
{
    union
    {
        float in;
        std::int32_t out;
    } data;
    data.in = input;
    std::bitset<sizeof(float) * CHAR_BIT> bits(data.out);
    const unsigned int mantissa_bits_to_truncate = 10;
    unsigned int token_exponent = 0;
    std::uint32_t token = 0;
    for(unsigned int i = 31; i > mantissa_bits_to_truncate; i--)
    {
        if(bits[i])
        {
            token += (1 << token_exponent);
        }
        token_exponent++;
    }
    return token;
}

float token_to_float(std::uint32_t input)
{
    std::bitset<sizeof(std::uint32_t) * CHAR_BIT> token_bits(input);
    std::bitset<sizeof(float) * CHAR_BIT> float_bits(0.0f);
    unsigned int float_bits_offset = 0;
    for(int i = 31; i > -1; i--)
    {
        float_bits[float_bits_offset++] = token_bits[i];
    }
    float output;
    memcpy(&output, &float_bits, sizeof(float));
    return output;
}*/

void launch_kernel(
    hipStream_t stream, void** buffers, const char* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc = 
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);
    std::cout << "truncate amount: " << desc.mantissa_bits_to_truncate << "\n";
}

}
