#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include "serde.h"
#include "tokenization.h"

namespace fp_tokenization
{

__global__ void fp32_to_token_kernel(float* input, uint32_t* output, uint32_t size) 
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < size) 
    {
        __half inter = __float2half(input[idx]);
        output[idx] = static_cast<uint32_t>(reinterpret_cast<uint16_t&>(inter));
    }
}

__global__ void token_to_fp32_kernel(uint32_t* input, float* output, uint32_t size) 
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < size) 
    {
        uint16_t inter = static_cast<uint16_t>(input[idx]);
        output[idx] = __half2float(reinterpret_cast<__half&>(inter));
    }
}

void fp32_to_token(
    hipStream_t stream, void** buffers, char const* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc =
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);

    float* input = static_cast<float*>(buffers[0]);
    uint32_t* output = static_cast<uint32_t*>(buffers[1]);
    const uint32_t size = desc.n_elements;
    const int threads_per_block = 256;
    const int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    fp32_to_token_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(input, output, size); 
}

void token_to_fp32(
    hipStream_t stream, void** buffers, char const* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc =
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);

    uint32_t* input = static_cast<uint32_t*>(buffers[0]);
    float* output = static_cast<float*>(buffers[1]);
    const uint32_t size = desc.n_elements;
    const int threads_per_block = 256;
    const int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    token_to_fp32_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(input, output, size); 
}

uint32_t get_fp32_to_token_vocab_size()
{
    // Any permutation of 16 bits is a valid token.
    constexpr uint32_t vocab_size = 1ULL << 16;  // 2^16
    return vocab_size;
}

__global__ void fp32_to_bitfield16_kernel(float* input, uint32_t* output, uint32_t size)
{
    constexpr uint32_t bitfield_size = 16;
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < size) 
    {
        __half inter = __float2half(input[idx]);
        uint16_t token = reinterpret_cast<uint16_t&>(inter);
        for(int k = 0; k < bitfield_size; ++k)
        {
            uint16_t mask_result = (token & (1 << k)) > 0 ? 1 : 0;
            output[idx*bitfield_size + k] = (uint32_t)mask_result;
        }
    }
}

__global__ void bitfield16_to_fp32_kernel(uint32_t* input, float* output, uint32_t size)
{
    constexpr uint32_t bitfield_size = 16;
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < size) 
    {
        uint16_t token = 0;
        for(uint16_t k = 0; k < bitfield_size; ++k)
        {
            uint32_t current_bit = input[idx*bitfield_size + k];
            
            if(current_bit != 0)
            {
                token += (uint16_t)(1 << k);
            }
        }
        float fp_result = __half2float(reinterpret_cast<__half&>(token));
        output[idx] = fp_result;
    }
}

void fp32_to_bitfield16(
    hipStream_t stream, void** buffers, char const* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc =
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);

    float* input = static_cast<float*>(buffers[0]);
    uint32_t* output = static_cast<uint32_t*>(buffers[1]);
    const uint32_t size = desc.n_elements;
    const int threads_per_block = 256;
    const int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    fp32_to_bitfield16_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(input, output, size); 
}

void bitfield16_to_fp32(
    hipStream_t stream, void** buffers, char const* opaque, std::size_t opaque_len
){
     tokenization_descriptor_t const &desc =
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);

    uint32_t* input = static_cast<uint32_t*>(buffers[0]);
    float* output = static_cast<float*>(buffers[1]);
    const uint32_t size = desc.n_elements;
    const int threads_per_block = 256;
    const int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    bitfield16_to_fp32_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(input, output, size); 
}

__global__ void fp32_to_u8_token_kernel(const float* input, uint8_t* output, uint32_t size)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        output[idx] = (uint8_t)__hip_cvt_float_to_fp8(input[idx], __HIP_SATFINITE, __HIP_E4M3_FNUZ);
    }
}

__global__ void u8_token_to_fp32_kernel(const uint8_t* input, float* output, uint32_t size)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        __half_raw inter = __hip_cvt_fp8_to_halfraw(input[idx], __HIP_E4M3_FNUZ);
        output[idx] = __half2float(inter);
    }
}

void fp32_to_u8_token(
    hipStream_t stream, void** buffers, char const* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc =
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);

    const float* input = static_cast<float*>(buffers[0]);
    uint8_t* output = static_cast<uint8_t*>(buffers[1]);
    const uint32_t size = desc.n_elements;
    const int threads_per_block = 256;
    const int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    fp32_to_u8_token_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(input, output, size); 
}

void u8_token_to_fp32(
    hipStream_t stream, void** buffers, char const* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc =
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);

    const uint8_t* input = static_cast<uint8_t*>(buffers[0]);
    float* output = static_cast<float*>(buffers[1]);
    const uint32_t size = desc.n_elements;
    const int threads_per_block = 256;
    const int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    u8_token_to_fp32_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(input, output, size); 
}

uint32_t get_fp32_to_u8_token_vocab_size()
{
    // Any permutation of 8 bits is a valid token.
    constexpr uint32_t vocab_size = 1ULL << 8;  // 2^8
    return vocab_size;
}

__global__ void fp32_to_byte_pair_token_kernel(const float* input, uint8_t* output, uint32_t size)
{
    constexpr uint16_t bf16_mantissa_size = 7;
    constexpr uint16_t bf16_exponent_size = 8;
    constexpr uint16_t bf16_sign_position = bf16_mantissa_size + bf16_exponent_size;
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size) 
    {
        __hip_bfloat16 inter = __float2bfloat16(input[idx]);
        const uint16_t unified_token = reinterpret_cast<uint16_t&>(inter);
        uint8_t mantissa_token = 0;
        uint8_t exponent_token = 0;
        
        // Add mantissa bits to mantissa token.
        for(uint16_t i = 0; i < bf16_mantissa_size; ++i)
        {
            mantissa_token += unified_token & (1u << i);
        }
        // Add exponent bits to exponent token.
        for(uint16_t i = bf16_mantissa_size; i < bf16_sign_position; ++i)
        {
            exponent_token += (unified_token & (1u << i)) >> bf16_mantissa_size;
        }
        // Add sign bit to mantissa token.
        mantissa_token += (unified_token & (1u << bf16_sign_position)) >> bf16_exponent_size;
        
        const uint32_t out_idx = idx * 2;
        output[out_idx] = mantissa_token;
        output[out_idx + 1] = exponent_token;
    }
}

__global__ void byte_pair_token_to_fp32_kernel(const uint8_t* input, float* output, uint32_t size)
{
    constexpr uint16_t bf16_mantissa_size = 7;
    constexpr uint16_t bf16_exponent_size = 8;
    constexpr uint16_t bf16_sign_position = bf16_mantissa_size + bf16_exponent_size;
    const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        const uint32_t in_idx = idx * 2;
        const uint8_t mantissa_token = input[in_idx];
        const uint8_t exponent_token = input[in_idx + 1];
        uint16_t unified_token = 0;

        // Merge mantissa bits.
        for(uint16_t i = 0; i < bf16_mantissa_size; ++i)
        {
            unified_token += static_cast<uint16_t>(mantissa_token & (1u << i));
        }
        // Merge exponent bits.
        for(uint16_t i = 0; i < bf16_exponent_size; ++i)
        {
            unified_token += static_cast<uint16_t>(exponent_token & (1u << i)) << bf16_mantissa_size;
        }
        // Merge sign bit.
        unified_token += static_cast<uint16_t>(mantissa_token & (1u << bf16_mantissa_size)) << bf16_exponent_size;

        __hip_bfloat16 inter = reinterpret_cast<__hip_bfloat16&>(unified_token);
        output[idx] = __bfloat162float(inter);;
    }
}

void fp32_to_byte_pair_token(
    hipStream_t stream, void** buffers, char const* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc =
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);

    const float* input = static_cast<float*>(buffers[0]);
    uint8_t* output = static_cast<uint8_t*>(buffers[1]);
    const uint32_t size = desc.n_elements;
    const int threads_per_block = 256;
    const int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    fp32_to_byte_pair_token_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(input, output, size); 
}

void byte_pair_token_to_fp32(
    hipStream_t stream, void** buffers, char const* opaque, std::size_t opaque_len
){
    tokenization_descriptor_t const &desc =
        *deserialize<tokenization_descriptor_t>(opaque, opaque_len);

    const uint8_t* input = static_cast<uint8_t*>(buffers[0]);
    float* output = static_cast<float*>(buffers[1]);
    const uint32_t size = desc.n_elements;
    const int threads_per_block = 256;
    const int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    byte_pair_token_to_fp32_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(input, output, size); 
}

//#define STANDALONE_PROGRAM
#ifdef STANDALONE_PROGRAM
int main()
{
    int size = 512;
    int threads_per_block = 256;
    int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    /* Tokenization. */
    size_t input_size_bytes = sizeof(float) * size;
    size_t output_size_bytes = sizeof(uint32_t) * size;
    float* h_input;
    uint32_t* h_output;
    float* d_input;
    uint32_t* d_output;
    h_input = (float*)malloc(input_size_bytes);
    h_output = (uint32_t*)malloc(output_size_bytes);
    hipMalloc(&d_input, input_size_bytes);
    hipMalloc(&d_output, output_size_bytes);

    printf("Input:\n");
    for(int i = 0; i < size; ++i)
    {
        h_input[i] = 4.0f * sinf(2.0f * 3.14159f * ((float)i / (float)size));
        printf("%f ", h_input[i]);
    }
    printf("\n");
    
    hipMemcpy(d_input, h_input, input_size_bytes, hipMemcpyHostToDevice);
        
    fp32_to_token_kernel<<<blocks_per_grid, threads_per_block>>>(d_input, d_output, size);
    
    hipError_t cuda_status;
    cuda_status = hipGetLastError();
    if(cuda_status != hipSuccess) 
    {
        fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cuda_status));
    }
    hipMemcpy(h_output, d_output, output_size_bytes, hipMemcpyDeviceToHost);
    printf("\nToken Output:\n");
    for(int i = 0; i < size; ++i)
    {
        printf("%u ", h_output[i]);
    }
    printf("\n");
    
    /* Detokenization. */
    float* h_output_fp32;
    float* d_output_fp32;
    h_output_fp32 = (float*)malloc(input_size_bytes);
    hipMalloc(&d_output_fp32, input_size_bytes);

    token_to_fp32_kernel<<<blocks_per_grid, threads_per_block>>>(d_output, d_output_fp32, size);
    
    cuda_status = hipGetLastError();
    if(cuda_status != hipSuccess) 
    {
        fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cuda_status));
    }
    hipMemcpy(h_output_fp32, d_output_fp32, input_size_bytes, hipMemcpyDeviceToHost);
    printf("\nFloat Output:\n");
    for(int i = 0; i < size; ++i)
    {
        printf("%f ", h_output_fp32[i]);
    }
    printf("\n");
}
#endif // STANDALONE_PROGRAM

} // namespace fp_tokenization
